#include "NMath.cuh"

Tensor::Tensor(float in_value) : Tensor(0, { }, { in_value })
{
}

Tensor::Tensor(int in_size, vector<float> in_values) : Tensor(1, { in_size }, in_values)
{
}

Tensor::Tensor(int in_width, int in_height, vector<float> in_values) : Tensor(2, { in_width, in_height }, in_values)
{
}

Tensor::Tensor(int in_order, vector<int> in_dimensionSizes, vector<float> in_values)
{
	order = in_order;

	dimensionSizes = vector<int>(order);

	size = 1;

	for (int i = 0; i < in_order; i++)
	{
		dimensionSizes[i] = in_dimensionSizes[i];
		size *= in_dimensionSizes[i];
	}

	values = vector<float>(size);

	for (int i = 0; i < size; i++)
	{
		values[i] = in_values[i];
	}
}