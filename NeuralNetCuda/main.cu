#include "hip/hip_runtime.h"
#include "main.cuh"

string removeChracter(string i_text, char i_character)
{
    string text;

    for(int i = 0; i < i_text.size(); i ++)
    {
        if(i_text[i] != i_character)
        {
            text += i_text[i];
        }
    }

    return text;
}

int main()
{
    srand (static_cast <unsigned> (time(0)));

    BoundingBoxStorage allBoundingBoxes;
    allBoundingBoxes.getAllBoundingBoxes(32, 96, 2.0f, 512, 16, 32);

    int boundingBoxCount = allBoundingBoxes.allBoundingBoxes.size();

    cout << "Welcome to PRN" << endl;

    cout << "Found " << boundingBoxCount << " bounding boxes" << endl;

    vector<MapType> layerType = {Input, Convolutional, Pooling, Convolutional, Pooling, Convolutional, Pooling, Connected, Connected};
    vector<int> mapCount = {3, 8, 8, 16, 16, 32, 32, 1, 1};
    vector<int> reduction = {64, 3, 2, 4, 2, 3, 2, 100, 2};

    ConvolutionalNetwork net = ConvolutionalNetwork(layerType, mapCount, reduction);

    cout << "Enter path for image" << endl;

    string path;
    getline(cin, path);

    path = removeChracter(path, '"');

    DIR *dir;
    struct dirent *diread;
    vector<string> fileNames;

    if ((dir = opendir(path.c_str())) != nullptr) 
    {
        while ((diread = readdir(dir)) != nullptr) 
        {
            fileNames.push_back(string(diread->d_name));
        }
        closedir (dir);
    } 
    else 
    {
        perror ("opendir");
        return EXIT_FAILURE;
    }

    for(int i = 0; i < fileNames.size(); i ++)
    {
        string fileString = fileNames[i];

        if(fileString[0] == '.')
            continue;

        string imagePath = path + "\\" + fileString;
        
        Image image = Image(imagePath, 1024);

        if(image.trafficSigns.size() == 0)
            continue;

        vector<Frame> subFrames = image.getSubImageFrames({ 32,48,64,96,128 }, { 1.0 }, 8);

        int bestIndex = 0;
        float bestObjectPercentage = 0;

        vector<Frame> positiveSamples = vector<Frame> ();
        vector<Frame> negativeSamples = vector<Frame>();
        for (int j = 0; j < subFrames.size(); j++)
        {
            if (subFrames[j].percentage > bestObjectPercentage)
            {
                bestIndex = j;
                bestObjectPercentage = subFrames[j].percentage;
            }

            if (subFrames[j].percentage > 0.7)
            {
                positiveSamples.push_back(subFrames[j]);

            }
            else if(subFrames[j].percentage == 0)
            {
                negativeSamples.push_back(subFrames[j]);
            }
        }

        int sampleSize = min(positiveSamples.size(), negativeSamples.size());

        if (sampleSize <= 0)
        {
            continue;
        }

        image.loadImage(1024);

        float correct = 0;

        for (int j = 0; j < sampleSize; j++)
        {             
            int positiveIndex = rand() % positiveSamples.size();
            Image positiveSample = image.getSubImage(positiveSamples[positiveIndex]);
            positiveSample.resizeImage(64);

            //positiveSample.printImage(j);

            vector<float> positiveObjectPercentage = { 1, 0 };
            net.Train(positiveSample.getImageData(), &positiveObjectPercentage);
            
            //cout << (*net.GetValues(net.layerCount - 1)[0])[0] << ", " << (*net.GetValues(net.layerCount - 1)[0])[1] << endl;

            vector<vector<float>*> inputNet = net.layers[0].GetValues();

            correct += pow(1 - (*net.GetValues(net.layerCount - 1)[0])[0],2) + pow((*net.GetValues(net.layerCount - 1)[0])[1],2);

            positiveSamples.erase(positiveSamples.begin() + positiveIndex);


            int negativeIndex = rand() % negativeSamples.size();
            Image negativeSample = image.getSubImage(negativeSamples[negativeIndex]);
            negativeSample.resizeImage(64);

            vector<float> negativeObjectPercentage = { 0, 1 };
            net.Train(negativeSample.getImageData(), &negativeObjectPercentage);

            //cout << (*net.GetValues(net.layerCount - 1)[0])[0] << ", " << (*net.GetValues(net.layerCount - 1)[0])[1] << endl;

            correct += pow(1 - (*net.GetValues(net.layerCount - 1)[0])[1],2) + pow((*net.GetValues(net.layerCount - 1)[0])[0],2);


            negativeSamples.erase(negativeSamples.begin() + negativeIndex);
        }

        net.Improve();


        if (sampleSize > 0)
        {
            float rating = (float)correct / (float)(2 * sampleSize);
            cout << "Rating: " << rating << endl;

        }

        //image.resizeImage(128);
        //image.printImage(-1);

        image.setOptimalResults(&allBoundingBoxes);
            
        vector<float>* netOutput = net.GetValues(net.layerCount - 1)[0];
    }

    return 0;
}