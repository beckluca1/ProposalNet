#include "hip/hip_runtime.h"
#include "main.cuh"

string removeChracter(string i_text, char i_character)
{
    string text;

    for(int i = 0; i < i_text.size(); i ++)
    {
        if(i_text[i] != i_character)
        {
            text += i_text[i];
        }
    }

    return text;
}

int main()
{
    srand (static_cast <unsigned> (time(0)));

    BoundingBoxStorage allBoundingBoxes;
    allBoundingBoxes.getAllBoundingBoxes(32, 96, 2.0f, 512, 16, 32);

    int boundingBoxCount = allBoundingBoxes.allBoundingBoxes.size();

    cout << "Welcome to PRN" << endl;

    cout << "Found " << boundingBoxCount << " bounding boxes" << endl;

    vector<MapType> layerType = {Input, Convolutional, Pooling, Convolutional, Pooling, Convolutional, Pooling, Connected, Connected};
    vector<int> mapCount = {3, 8, 8, 16, 16, 32, 32, 1, 1};
    vector<int> reduction = {128, 3, 2, 4, 2, 3, 2, 100, 2};

    ConvolutionalNetwork net = ConvolutionalNetwork(layerType, mapCount, reduction);

    cout << "Enter path for image" << endl;

    string path;
    getline(cin, path);

    path = removeChracter(path, '"');

    DIR *dir;
    struct dirent *diread;
    vector<string> fileNames;

    if ((dir = opendir(path.c_str())) != nullptr) 
    {
        while ((diread = readdir(dir)) != nullptr) 
        {
            fileNames.push_back(string(diread->d_name));
        }
        closedir (dir);
    } 
    else 
    {
        perror ("opendir");
        return EXIT_FAILURE;
    }

    for(int i = 0; i < fileNames.size(); i ++)
    {
        string fileString = fileNames[i];

        if(fileString[0] == '.')
            continue;

        cout << "File: " << fileString << endl;

        string imagePath = path + "\\" + fileString;
        //cout << imagePath << endl;

        Image image = Image(imagePath, 1024);

        if(image.trafficSigns.size() == 0)
            continue;

        vector<Image> subImages = image.getSubImages({ 64,128,256 }, { 1.0 }, 64);

        int bestIndex = 0;
        float bestObjectPercentage = 0;

        vector<Image> positiveSamples = vector<Image> ();
        vector<Image> negativeSamples = vector<Image>();
        for (int j = 0; j < subImages.size(); j++)
        {
            if (subImages[j].objectPercentage > bestObjectPercentage)
            {
                bestIndex = j;
                bestObjectPercentage = subImages[j].objectPercentage;
            }

            if (subImages[j].objectPercentage > 0.3)
            {
                positiveSamples.push_back(subImages[j]);

            }
            else if(subImages[j].objectPercentage == 0)
            {
                negativeSamples.push_back(subImages[j]);
            }
        }

        image = subImages[bestIndex];

        int sampleSize = min(positiveSamples.size(), negativeSamples.size());

        float correct = 0;

        for (int j = 0; j < sampleSize; j++)
        {
            int positiveIndex = rand() % positiveSamples.size();

            positiveSamples[positiveIndex].resizeImage(128);
            vector<float> positiveObjectPercentage = { 1, 0 };// positiveSamples[positiveIndex]->objectPercentage};
            net.Train(positiveSamples[positiveIndex].getImageData(), &positiveObjectPercentage);

            vector<vector<float>*> inputNet = net.layers[0].GetValues();
            //cout << "Positive test: " << (*net.GetValues(net.layerCount - 1)[0])[0] << ", " << (*net.GetValues(net.layerCount - 1)[0])[1] << " / " << positiveSamples[positiveIndex].objectPercentage << endl;

            correct += 1 - (*net.GetValues(net.layerCount - 1)[0])[0] + (*net.GetValues(net.layerCount - 1)[0])[1];

            positiveSamples.erase(positiveSamples.begin() + positiveIndex);


            int negativeIndex = rand() % negativeSamples.size();

            negativeSamples[negativeIndex].resizeImage(128);
            vector<float> negativeObjectPercentage = { 0, 1 };
            net.Train(negativeSamples[negativeIndex].getImageData(), &negativeObjectPercentage);

            //cout << "Negative test: " << (*net.GetValues(net.layerCount - 1)[0])[0] << ", " << (*net.GetValues(net.layerCount - 1)[0])[1] << " / " << negativeSamples[negativeIndex].objectPercentage << endl;

            correct += 1 - (*net.GetValues(net.layerCount - 1)[0])[1] + (*net.GetValues(net.layerCount - 1)[0])[0];


            negativeSamples.erase(negativeSamples.begin() + negativeIndex);
        }

        net.Improve();


        if (sampleSize > 0)
        {
            float rating = (float)correct / (float)(2 * sampleSize);
            cout << "Rating: " << rating << endl;

        }

        image.resizeImage(128);

        //cout << "Get image data" << endl;


        //cout << "Rate bounding boxes" << endl;

        image.setOptimalResults(&allBoundingBoxes);
            
        //cout << "Train net" << endl;

        vector<float>* netOutput = net.GetValues(net.layerCount - 1)[0];
        //image.setBestNetResults(netOutput);

        //cout << "Improve net" << endl;

        if(image.objectPercentage > 0.3)
        {
            image.printImage();
        }
    }

    return 0;
}