#include "hip/hip_runtime.h"
#include "NeuralNet.cuh"

int ID;
float LEARN_RATE = 0.1f;
int BLOCKS = 1;
int THREADS = 1024;

__device__ float sigmoidCuda(float value)
{
    return (1.0f / (exp(-value) + 1.0f));
}

__device__ float d_sigmoidCuda(float value)
{
    float sig = sigmoidCuda(value);
    float d_sig = sig * (1 - sig);
    return d_sig;
}

__global__ void ResetActivationsZeroCuda(float* activations, int* threadOffset)
{
    int idx = threadIdx.x + *threadOffset;

    activations[idx] = 0;
}

__global__ void ResetActivationsCuda(float* activations, float* bias, int* threadOffset)
{
    int idx = threadIdx.x + *threadOffset;

    activations[idx] = bias[idx];
}

__global__ void UpdateConvolutionalMapCuda(float* activations, float* previousMapValues, float* kernelWeights, int* mapSize, int* kernelSize, int* previousMapSize, int* threadOffset)
{
    int idx = threadIdx.x + *threadOffset;

    int x = idx % *mapSize;
    int y = (idx - x) / *mapSize;

    int mapIndex = idx;

    for (int dY = 0; dY < *kernelSize; dY++)
    {
        for (int dX = 0; dX < *kernelSize; dX++)
        {
            int previousIndex = (x + dX) + *previousMapSize * (y + dY);
            int weightIndex = dX + *kernelSize * dY;

            activations[mapIndex] += previousMapValues[previousIndex] * kernelWeights[weightIndex];
        }
    }
}

__global__ void SetValuesCuda(float* activations, float* values, int* threadOffset)
{
    int idx = threadIdx.x + *threadOffset;

    values[idx] = sigmoidCuda(activations[idx]);
}

__global__ void CalculateChangesConvolutionalMapCuda(float* valueChanges, float* previousMapActivations, float* previousMapValues, float* previousMapValueChanges, float* kernelWeights, float* kernelWeightChanges, int* mapSize, int* kernelSize, int* previousMapSize, bool* pooling, int* threadOffset)
{
    int idx = threadIdx.x + *threadOffset;

    int x = idx % *mapSize;
    int y = (idx - x) / *mapSize;

    int mapIndex = idx;

    for (int dY = 0; dY < *kernelSize; dY++)
    {
        for (int dX = 0; dX < *kernelSize; dX++)
        {
            int previousIndex = (x + dX) + *previousMapSize * (y + dY);
            int weightIndex = dX + *kernelSize * dY;

            kernelWeightChanges[weightIndex] += previousMapValues[previousIndex] * valueChanges[mapIndex];

            if (*pooling)
            {
                previousMapValueChanges[previousIndex] += kernelWeights[weightIndex] * valueChanges[mapIndex];
            }
            else
            {
                previousMapValueChanges[previousIndex] += kernelWeights[weightIndex] * d_sigmoidCuda(previousMapActivations[previousIndex]) * valueChanges[mapIndex];
            }
        }
    }
}

__global__ void UpdatePoolingMapCuda(float* activations, float* values, float* previousMapValues, int* mapSize, int* poolingSize, int* previousMapSize, int* threadOffset)
{
    int idx = threadIdx.x + *threadOffset;

    int x = idx % *mapSize;
    int y = (idx - x) / *mapSize;

    int mapIndex = idx;
    int previousIndex = (x + *previousMapSize * y) * *poolingSize;

    float maximum = previousMapValues[previousIndex];
    for (int dY = 0; dY < *poolingSize; dY++)
    {
        for (int dX = 0; dX < *poolingSize; dX++)
        {
            previousIndex = (x * *poolingSize + dX) + *previousMapSize * (y * *poolingSize + dY);

            float value = previousMapValues[previousIndex];
            maximum = value > maximum ? value : maximum;
        }
    }

    activations[mapIndex] = maximum;
    values[mapIndex] = maximum;
}

__global__ void UpdateConnectedMapCuda(float* activations, float* previousMapValues, float* weights, int* mapSize, int* previousMapIndex, int* previousMapSize, int* previousMapCount, int* threadOffset)
{
    int idx = threadIdx.x + *threadOffset;

    int previousIndex = idx % *previousMapSize;
    int mapIndex = (idx - previousIndex) / *previousMapSize;
    int weightIndex = *previousMapIndex + *previousMapCount * mapIndex + *previousMapCount * *mapSize * previousIndex;

    activations[mapIndex] += previousMapValues[previousIndex] * weights[weightIndex];
}

__global__ void SetBiasCuda(float* valueChanges, float* biasWeightChanges, int* threadOffset)
{
    int idx = threadIdx.x + *threadOffset;

    biasWeightChanges[idx] += valueChanges[idx];
}

__global__ void CalculateChangesConnectedMapCuda(float* valueChanges, float* previousMapActivations, float* previousMapValues, float* previousMapValueChanges, float* weights, float* weightChanges, int* mapSize, int* previousMapIndex, int* previousMapSize, int* previousMapCount, bool* pooling, int* threadOffset)
{
    int idx = threadIdx.x + *threadOffset;

    int previousIndex = idx % *previousMapSize;
    int mapIndex = (idx - previousIndex) / *previousMapSize;
    int weightIndex = *previousMapIndex + *previousMapCount * mapIndex + *previousMapCount * *mapSize * previousIndex;

    weightChanges[weightIndex] += previousMapValues[previousIndex] * valueChanges[mapIndex];

    if (*pooling)
    {
        previousMapValueChanges[previousIndex] += weights[weightIndex] * valueChanges[mapIndex];

    }
    else
    {
        previousMapValueChanges[previousIndex] += weights[weightIndex] * d_sigmoidCuda(previousMapActivations[previousIndex]) * valueChanges[mapIndex];
    }
    
}

float sigmoid(float value)
{
    return (1.0f / (exp(-value) + 1.0f));
}

float d_sigmoid(float value)
{
    float sig = sigmoid(value);
    float d_sig = sig*(1-sig);
    return d_sig;
}

WeightStorage::WeightStorage()
{
}

WeightStorage::WeightStorage(int i_width, int i_height, float i_randomBounds)
{
    id = ID++;

    width = i_width;
    height = i_height;

    weightCount = width * height;

    for(int i = 0; i < weightCount; i++)
    {
        weights.push_back(static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (2*i_randomBounds))) - i_randomBounds);
        weightChanges.push_back(0.0f);
    }
}

void WeightStorage::Improve()
{
    for(int i = 0; i < weightCount; i++)
    {
        weights[i] -= LEARN_RATE * weightChanges[i];
        weightChanges[i] = 0;
    }
}

NeuralKernel::NeuralKernel(int i_kernelSize, float i_randomBounds) : WeightStorage(i_kernelSize, i_kernelSize, i_randomBounds)
{
    kernelSize = i_kernelSize;
}

NeuralMap::NeuralMap()
{
}

void NeuralMap::SetNodeCount(int i_nodeCount)
{
    nodeCount = i_nodeCount;

    activations = vector<float>(nodeCount);
    values = vector<float>(nodeCount);
    valueChanges = vector<float>(nodeCount);
}

void NeuralMap::SetValues(vector<float>* i_values)
{
    for(int i = 0; i < i_values->size(); i++)
    {
        values[i] = (*i_values)[i];
    }
}

vector<float>* NeuralMap::GetValues()
{
    return &values;
}

void NeuralMap::Correct(vector<float>* i_values)
{    
    for(int i = 0; i < (*i_values).size(); i ++)
    {
        if((*i_values)[i] != - 1)
        {
            valueChanges[i] = 2*(values[i] - (*i_values)[i]);
        }
    }   
}

void NeuralMap::PrintValues()
{
    cout << GetMapType() << " (id: " << id << ", size: " << mapSize <<  ") ";

    if(previousMaps.size() != 0)
    {
        cout << "[input id: ";
        for(int i = 0; i < previousMaps.size()-1; i ++)
        {
            cout << previousMaps[i]->id << ", ";
        }
        cout << previousMaps[previousMaps.size() - 1]->id << "]";
    }

    cout << endl;

    for(int i = 0; i < nodeCount; i++)
    {
        if(((i+1) % mapSize) == 0)
        {
            cout << fixed << setprecision(2) << values[i] << endl;
        }
        else
        {
            cout << fixed << setprecision(2)  << values[i] << " | ";
        }
    }
}

string NeuralMap::GetMapType()
{
    switch (mapType)
    {
        case Input:
            return "Input Map";
        case Convolutional:
            return "Convolutional Map";
        case Pooling:
            return "Pooling Map";
        case Connected:
            return "Connected Map";
        default:
            return "Other Map";
    }
}

InputMap::InputMap()
{
}

InputMap::InputMap(int i_mapSize)
{
    id = ID++;

    mapType = Input;
    mapSize = i_mapSize;

    SetNodeCount(i_mapSize * i_mapSize);
}

void InputMap::Update()
{
}

void InputMap::CalculateChanges()
{
}

void InputMap::Improve()
{
}

ConvolutionalMap::ConvolutionalMap()
{      
}

ConvolutionalMap::ConvolutionalMap(int i_kernelSize, NeuralMap* i_previousMap)
{
    id = ID++;

    mapType = Convolutional;

    kernelSize = i_kernelSize;

    previousMaps = {i_previousMap};
    mapSize = previousMaps[0]->mapSize - kernelSize + 1;

    SetNodeCount(mapSize * mapSize);
            
    for(int i = 0; i < previousMaps.size(); i ++)
    {
        kernels.push_back(NeuralKernel(kernelSize, 3.0f));
    }
}

ConvolutionalMap::ConvolutionalMap(int i_kernelSize, vector<NeuralMap*> i_previousMaps)
{      
    id = ID++;

    mapType = Convolutional;

    kernelSize = i_kernelSize;

    previousMaps = i_previousMaps;
    mapSize = previousMaps[0]->mapSize - kernelSize + 1;

    SetNodeCount(mapSize * mapSize);
            
    for(int i = 0; i < previousMaps.size(); i ++)
    {
        kernels.push_back(NeuralKernel(kernelSize, 3.0f));
    }
}

void ConvolutionalMap::Update()
{
    int previousMapCount = previousMaps.size();
    int previousMapSize = previousMaps[0]->mapSize;

    hipSetDevice(0);

    int* dev_threadOffset = 0;
    
    float* dev_activations = 0;
    float* dev_values = 0;

    float* dev_previousMapValues = 0;
    float* dev_kernelWeights = 0;

    int* dev_mapSize = 0;
    int* dev_previousMapSize = 0;
    int* dev_kernelSize = 0;

    hipMalloc((void**)&dev_threadOffset, 1 * sizeof(int));

    hipMalloc((void**)&dev_activations, mapSize * mapSize * sizeof(float));
    hipMalloc((void**)&dev_values, mapSize * mapSize * sizeof(float));

    hipMalloc((void**)&dev_mapSize, 1 * sizeof(int));
    hipMalloc((void**)&dev_previousMapSize, 1 * sizeof(int));
    hipMalloc((void**)&dev_kernelSize, 1 * sizeof(int));

    hipMalloc((void**)&dev_previousMapValues, previousMapSize * previousMapSize * sizeof(float));
    hipMalloc((void**)&dev_kernelWeights, kernelSize * kernelSize * sizeof(float));

    hipMemcpy(dev_activations, &activations[0], mapSize * mapSize * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(dev_mapSize, &mapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_previousMapSize, &previousMapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_kernelSize, &kernelSize, 1 * sizeof(int), hipMemcpyHostToDevice);

    int allThreads = mapSize * mapSize;

    for (int i = 0; i < allThreads; i += THREADS)
    {
        int threadOffset = i;

        hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

        ResetActivationsZeroCuda << <BLOCKS, min(THREADS, allThreads - i) >> > (dev_activations, dev_threadOffset);
    }

    hipDeviceSynchronize();
    
    for(int i = 0; i < previousMapCount; i ++)
    {
        NeuralMap* previousMap = previousMaps[i];

        NeuralKernel* kernel = &kernels[i];

        hipMemcpy(dev_previousMapValues, &previousMap->values[0], previousMapSize * previousMapSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_kernelWeights, &kernel->weights[0], kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);
        
        for (int j = 0; j < allThreads; j += THREADS)
        {
            int threadOffset = j;
            
            hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

            UpdateConvolutionalMapCuda << <BLOCKS, min(THREADS, allThreads - j) >> > (dev_activations, dev_previousMapValues, dev_kernelWeights, dev_mapSize, dev_kernelSize, dev_previousMapSize, dev_threadOffset);
        }
    }

    hipDeviceSynchronize();

    for (int i = 0; i < allThreads; i += THREADS)
    {
        int threadOffset = i;

        hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

        SetValuesCuda << <BLOCKS, min(THREADS, allThreads - i) >> > (dev_activations, dev_values, dev_threadOffset);
    }

    hipDeviceSynchronize();

    hipMemcpy(&activations[0], dev_activations, mapSize * mapSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&values[0], dev_values, mapSize * mapSize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_threadOffset);

    hipFree(dev_activations);
    hipFree(dev_values);

    hipFree(dev_previousMapValues);
    hipFree(dev_kernelWeights);

    hipFree(dev_mapSize);
    hipFree(dev_previousMapSize);
    hipFree(dev_kernelSize);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ConvolutionalMap::Update failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

void ConvolutionalMap::CalculateChanges()
{
    int previousMapCount = previousMaps.size();
    int previousMapSize = previousMaps[0]->mapSize;

    bool pooling = previousMaps[0]->mapType == Pooling;

    hipSetDevice(0);

    int* dev_threadOffset = 0;

    float* dev_valueChanges = 0;
    float* dev_previousMapActivations = 0;
    float* dev_previousMapValues = 0;
    float* dev_previousMapValueChanges = 0;
    float* dev_kernelWeights = 0;
    float* dev_kernelWeightChanges = 0;

    int* dev_kernelSize = 0;
    int* dev_previousMapSize = 0;
    int* dev_mapSize = 0;

    bool* dev_pooling = false;

    hipMalloc(&dev_threadOffset, 1 * sizeof(int));
    
    hipMalloc((void**)&dev_valueChanges, mapSize * mapSize * sizeof(float));

    hipMalloc((void**)&dev_previousMapActivations, previousMapSize * previousMapSize * sizeof(float));
    hipMalloc((void**)&dev_previousMapValues, previousMapSize * previousMapSize * sizeof(float));
    hipMalloc((void**)&dev_previousMapValueChanges, previousMapSize * previousMapSize * sizeof(float));
    hipMalloc((void**)&dev_kernelWeights, kernelSize * kernelSize * sizeof(float));
    hipMalloc((void**)&dev_kernelWeightChanges, kernelSize * kernelSize * sizeof(float));

    hipMalloc((void**)&dev_previousMapSize, 1 * sizeof(int));
    hipMalloc((void**)&dev_mapSize, 1 * sizeof(int));
    hipMalloc((void**)&dev_kernelSize, 1 * sizeof(int));

    hipMalloc((void**)&dev_pooling, 1 * sizeof(bool));

    hipMemcpy(dev_valueChanges, &valueChanges[0], mapSize * mapSize * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(dev_mapSize, &mapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_previousMapSize, &previousMapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_kernelSize, &kernelSize, 1 * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(dev_pooling, &pooling, 1 * sizeof(bool), hipMemcpyHostToDevice);

    for(int i = 0; i < previousMapCount; i ++)
    {
        NeuralMap* previousMap = previousMaps[i];

        NeuralKernel* kernel = &kernels[i];

        hipMemcpy(dev_previousMapActivations, &previousMap->activations[0], previousMapSize * previousMapSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_previousMapValues, &previousMap->values[0], previousMapSize * previousMapSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_previousMapValueChanges, &previousMap->valueChanges[0], previousMapSize * previousMapSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_kernelWeights, &kernel->weights[0], kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_kernelWeightChanges, &kernel->weightChanges[0], kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);
        
        int allThreads = mapSize * mapSize;

        for (int j = 0; j < allThreads; j += THREADS)
        {
            int threadOffset = j;

            hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

            CalculateChangesConvolutionalMapCuda << <BLOCKS, min(THREADS, allThreads - j) >> > (dev_valueChanges, dev_previousMapActivations, dev_previousMapValues, dev_previousMapValueChanges, dev_kernelWeights, dev_kernelWeightChanges, dev_mapSize, dev_kernelSize, dev_previousMapSize, dev_pooling, dev_threadOffset);
        }

        hipDeviceSynchronize();

        hipMemcpy(&kernel->weightChanges[0], dev_kernelWeightChanges, kernelSize * kernelSize * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&previousMap->valueChanges[0], dev_previousMapValueChanges, previousMapSize * previousMapSize * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipFree(dev_threadOffset);

    hipFree(dev_valueChanges);
    hipFree(dev_previousMapActivations);
    hipFree(dev_previousMapValues);
    hipFree(dev_previousMapValueChanges);
    hipFree(dev_kernelWeights);
    hipFree(dev_kernelWeightChanges);

    hipFree(dev_kernelSize);
    hipFree(dev_previousMapSize);
    hipFree(dev_mapSize);

    hipFree(dev_pooling);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ConvolutionalMap::CalculateChanges failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

void ConvolutionalMap::Improve()
{
    for(int i = 0; i < kernels.size(); i ++)
    {
        kernels[i].Improve();
    }

    for(int i = 0; i < nodeCount; i++)
    {
        valueChanges[i] = 0;
    }     
}

PoolingMap::PoolingMap()
{
}

PoolingMap::PoolingMap(int i_poolingSize, NeuralMap* i_previousMap)
{
    id = ID++;

    mapType = Pooling;

    previousMaps = {i_previousMap};

    poolingSize = i_poolingSize;
    mapSize = previousMaps[0]->mapSize / poolingSize;

    SetNodeCount(mapSize * mapSize);
}

PoolingMap::PoolingMap(int i_poolingSize, vector<NeuralMap*> i_previousMaps)
{
    id = ID++;

    mapType = Pooling;

    previousMaps = i_previousMaps;

    poolingSize = i_poolingSize;
    mapSize = previousMaps[0]->mapSize / poolingSize;

    SetNodeCount(mapSize * mapSize);
}

void PoolingMap::Update()
{
    int previousMapCount = previousMaps.size();
    int previousMapSize = previousMaps[0]->mapSize;

    hipSetDevice(0);

    int* dev_threadOffset = 0;

    float* dev_activations = 0;
    float* dev_values = 0;
    float* dev_previousMapValues = 0;

    int* dev_poolingSize = 0;
    int* dev_previousMapSize = 0;
    int* dev_mapSize = 0;

    hipMalloc(&dev_threadOffset, 1 * sizeof(int));

    hipMalloc((void**)&dev_activations, mapSize * mapSize * sizeof(float));
    hipMalloc((void**)&dev_values, mapSize * mapSize * sizeof(float));
    hipMalloc((void**)&dev_previousMapValues, previousMapSize * previousMapSize * sizeof(float));

    hipMalloc((void**)&dev_previousMapSize, 1 * sizeof(int));
    hipMalloc((void**)&dev_mapSize, 1 * sizeof(int));
    hipMalloc((void**)&dev_poolingSize, 1 * sizeof(int));

    hipMemcpy(dev_mapSize, &mapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_previousMapSize, &previousMapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_poolingSize, &poolingSize, 1 * sizeof(int), hipMemcpyHostToDevice);

    for(int i = 0; i < previousMapCount; i ++)
    {
        NeuralMap* previousMap = previousMaps[i];

        hipMemcpy(dev_previousMapValues, &previousMap->values[0], previousMapSize * previousMapSize * sizeof(float), hipMemcpyHostToDevice);

        int allThreads = mapSize * mapSize;

        for (int j = 0; j < allThreads; j += THREADS)
        {
            int threadOffset = j;

            hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

            UpdatePoolingMapCuda << <BLOCKS, min(THREADS, allThreads - j) >> > (dev_activations, dev_values, dev_previousMapValues, dev_mapSize, dev_poolingSize, dev_previousMapSize, dev_threadOffset);
        }

        hipDeviceSynchronize();
    }

    hipMemcpy(&activations[0], dev_activations, mapSize * mapSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&values[0], dev_values, mapSize * mapSize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_values);
    hipFree(dev_previousMapValues);

    hipFree(dev_previousMapSize);
    hipFree(dev_mapSize);
    hipFree(dev_poolingSize);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "PoolingMap::Update failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

void PoolingMap::CalculateChanges()
{
    for(int i = 0; i < previousMaps.size(); i ++)
        {
        NeuralMap* previousMap = previousMaps[i];
        int previousMapSize = previousMap->mapSize;

        for(int y = 0; y < mapSize; y ++)
        {
            for(int x = 0; x < mapSize; x ++)
            {
                int mapIndex = x + mapSize * y;
                int previousIndex = (x + previousMapSize * y) * poolingSize;
                
                float maximum = previousMap->values[previousIndex];
                int maximumIndex = previousIndex;

                for(int dY = 0; dY < poolingSize; dY ++)
                {
                    for(int dX = 0; dX < poolingSize; dX ++)
                    {
                        float value = previousMap->values[previousIndex];

                        maximumIndex = value > maximum ? previousIndex : maximumIndex;
                        maximum = value > maximum ? value : maximum;
                    }
                }

                previousMap->valueChanges[maximumIndex] += valueChanges[mapIndex] * d_sigmoid(previousMap->activations[maximumIndex]);
            }
        }
    }
}

void PoolingMap::Improve()
{
    for(int i = 0; i < nodeCount; i++)
    {
        valueChanges[i] = 0;
    }
}

ConnectedMap::ConnectedMap()
{
}

ConnectedMap::ConnectedMap(int i_nodeCount, NeuralMap* i_previousMap)
{
    id = ID++;

    mapType = Connected;
    mapSize = i_nodeCount;

    SetNodeCount(i_nodeCount);

    previousMaps = {i_previousMap};

    bias = WeightStorage(mapSize, 1, 1.0f);
    weights = WeightStorage(mapSize, previousMaps[0]->mapSize, 1.0f);

    weightCount = weights.weightCount;
}

ConnectedMap::ConnectedMap(int i_nodeCount, vector<NeuralMap*> i_previousMaps)
{
    id = ID++;

    mapType = Connected;
    mapSize = i_nodeCount;

    SetNodeCount(i_nodeCount);

    previousMaps = i_previousMaps;

    bias = WeightStorage(mapSize, 1, 1.0f);
    weights = WeightStorage(mapSize, previousMaps.size() * previousMaps[0]->mapSize, 1.0f);

    weightCount = weights.weightCount;
}

void ConnectedMap::Update()
{
    int previousMapCount = previousMaps.size();
    int previousMapSize = previousMaps[0]->mapSize;

    hipSetDevice(0);

    int* dev_threadOffset = 0;

    float* dev_activations = 0;
    float* dev_values = 0;
    float* dev_bias = 0;
    float* dev_weights = 0;
    float* dev_previousMapValues = 0;

    int* dev_mapSize = 0;
    int* dev_previousMapCount = 0;

    int* dev_previousMapIndex = 0;
    int* dev_previousMapSize = 0;

    hipMalloc((void**)&dev_threadOffset, 1 * sizeof(int));

    hipMalloc((void**)&dev_activations, mapSize * sizeof(float));
    hipMalloc((void**)&dev_values, mapSize * sizeof(float));
    hipMalloc((void**)&dev_bias, mapSize * sizeof(float));
    hipMalloc((void**)&dev_weights, mapSize * previousMapSize * previousMapCount * sizeof(float));
    hipMalloc((void**)&dev_previousMapValues, previousMapSize * sizeof(float));

    hipMalloc((void**)&dev_mapSize, 1 * sizeof(int));
    hipMalloc((void**)&dev_previousMapCount, 1 * sizeof(int));

    hipMalloc((void**)&dev_previousMapIndex, 1 * sizeof(int));
    hipMalloc((void**)&dev_previousMapSize, 1 * sizeof(int));

    hipMemcpy(dev_activations, &activations[0], mapSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_bias, &bias.weights[0], mapSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_weights, &weights.weights[0], mapSize * previousMapSize * previousMapCount * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(dev_mapSize, &mapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_previousMapSize, &previousMapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_previousMapCount, &previousMapCount, 1 * sizeof(int), hipMemcpyHostToDevice);

    int allThreads = mapSize;

    for (int i = 0; i < allThreads; i += THREADS)
    {
        int threadOffset = i;

        hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

        ResetActivationsCuda << <BLOCKS, min(THREADS, allThreads - i) >> > (dev_activations, dev_bias, dev_threadOffset);
    }

    hipDeviceSynchronize();

    for(int i = 0; i < previousMaps.size(); i ++)
    {
        int previousMapIndex = i;
        
        NeuralMap* previousMap = previousMaps[previousMapIndex];

        hipMemcpy(dev_previousMapValues, &previousMap->values[0], previousMapSize * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(dev_previousMapIndex, &previousMapIndex, 1 * sizeof(int), hipMemcpyHostToDevice);

        allThreads = mapSize * previousMapSize;

        for (int i = 0; i < allThreads; i += THREADS)
        {
            int threadOffset = i;

            hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

            UpdateConnectedMapCuda << <BLOCKS, min(THREADS, allThreads - i) >> > (dev_activations, dev_previousMapValues, dev_weights, dev_mapSize, dev_previousMapIndex, dev_previousMapSize, dev_previousMapCount, dev_threadOffset);
        }
    }

    hipDeviceSynchronize();

    allThreads = mapSize;

    for (int i = 0; i < allThreads; i += THREADS)
    {
        int threadOffset = i;

        hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

        SetValuesCuda << <BLOCKS, min(THREADS, allThreads - i) >> > (dev_activations, dev_values, dev_threadOffset);
    }

    hipDeviceSynchronize();

    hipMemcpy(&activations[0], dev_activations, mapSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&values[0], dev_values, mapSize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_threadOffset);
    
    hipFree(dev_activations);
    hipFree(dev_values);

    hipFree(dev_previousMapValues);
    hipFree(dev_weights);

    hipFree(dev_previousMapIndex);
    hipFree(dev_previousMapSize);
    hipFree(dev_bias);

    hipFree(dev_mapSize);
    hipFree(dev_previousMapCount);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ConnectedMap::Update failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

void ConnectedMap::CalculateChanges()
{
    int previousMapCount = previousMaps.size();
    int previousMapSize = previousMaps[0]->mapSize;

    bool pooling = previousMaps[0]->mapType == Pooling;

    hipSetDevice(0);

    int* dev_threadOffset = 0;

    float* dev_activations = 0;
    float* dev_valueChanges = 0;
    float* dev_bias = 0;
    float* dev_biasChanges = 0;
    float* dev_weights = 0;
    float* dev_weightChanges = 0;
    float* dev_previousMapActivations = 0;
    float* dev_previousMapValues = 0;
    float* dev_previousMapValueChanges = 0;

    int* dev_mapSize = 0;
    int* dev_previousMapCount = 0;

    int* dev_previousMapIndex = 0;
    int* dev_previousMapSize = 0;

    bool* dev_pooling = 0;

    hipMalloc((void**)&dev_threadOffset, 1 * sizeof(int));

    hipMalloc((void**)&dev_activations, mapSize * sizeof(float));
    hipMalloc((void**)&dev_valueChanges, mapSize * sizeof(float));
    hipMalloc((void**)&dev_bias, mapSize * sizeof(float));
    hipMalloc((void**)&dev_biasChanges, mapSize * sizeof(float));
    hipMalloc((void**)&dev_weights, mapSize * previousMapSize * previousMaps.size() * sizeof(float));
    hipMalloc((void**)&dev_weightChanges, mapSize * previousMapSize * previousMaps.size() * sizeof(float));
    hipMalloc((void**)&dev_previousMapActivations, previousMapSize * sizeof(float));
    hipMalloc((void**)&dev_previousMapValues, previousMapSize * sizeof(float));
    hipMalloc((void**)&dev_previousMapValueChanges, previousMapSize * sizeof(float));

    hipMalloc((void**)&dev_mapSize, 1 * sizeof(int));
    hipMalloc((void**)&dev_previousMapCount, 1 * sizeof(int));

    hipMalloc((void**)&dev_previousMapIndex, 1 * sizeof(int));
    hipMalloc((void**)&dev_previousMapSize, 1 * sizeof(int));

    hipMalloc((void**)&dev_pooling, 1 * sizeof(bool));

    hipMemcpy(dev_activations, &activations[0], mapSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_valueChanges, &valueChanges[0], mapSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_bias, &bias.weights[0], mapSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_biasChanges, &bias.weightChanges[0], mapSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_weights, &weights.weights[0], mapSize * previousMapSize * previousMapCount * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_weightChanges, &weights.weightChanges[0], mapSize * previousMapSize * previousMapCount * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(dev_mapSize, &mapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_previousMapSize, &previousMapSize, 1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_previousMapCount, &previousMapCount, 1 * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(dev_pooling, &pooling, 1 * sizeof(bool), hipMemcpyHostToDevice);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ConnectedMap::CalculateChanges SetUp failed: %s\n", hipGetErrorString(cudaStatus));
    }

    int allThreads = mapSize;

    for (int i = 0; i < allThreads; i += THREADS)
    {
        int threadOffset = i;

        hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

        SetBiasCuda << <BLOCKS, min(THREADS, allThreads - i) >> > (dev_valueChanges, dev_biasChanges, dev_threadOffset);
    }

    hipDeviceSynchronize();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ConnectedMap::CalculateChanges Bias failed: %s\n", hipGetErrorString(cudaStatus));
    }

    for(int i = 0; i < previousMapCount; i ++)
    {
        int previousMapIndex = i;

        NeuralMap* previousMap = previousMaps[previousMapIndex];

        hipMemcpy(dev_previousMapActivations, &previousMap->activations[0], previousMapSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_previousMapValues, &previousMap->values[0], previousMapSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_previousMapValueChanges, &previousMap->valueChanges[0], previousMapSize * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(dev_previousMapIndex, &previousMapIndex, 1 * sizeof(int), hipMemcpyHostToDevice);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "ConnectedMap::CalculateChanges Changes Memcpy failed: %s\n", hipGetErrorString(cudaStatus));
        }

        allThreads = mapSize * previousMapSize;

        for (int j = 0; j < allThreads; j += THREADS)
        {
            int threadOffset = j;

            hipMemcpy(dev_threadOffset, &threadOffset, 1 * sizeof(int), hipMemcpyHostToDevice);

            CalculateChangesConnectedMapCuda << <BLOCKS, min(THREADS, allThreads - j) >> > (dev_valueChanges, dev_previousMapActivations, dev_previousMapValues, dev_previousMapValueChanges, dev_weights, dev_weightChanges, dev_mapSize, dev_previousMapIndex, dev_previousMapSize, dev_previousMapCount, dev_pooling, dev_threadOffset);
        }

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "ConnectedMap::CalculateChanges Changes Run failed: %s\n", hipGetErrorString(cudaStatus));
        }

        hipDeviceSynchronize();

        hipMemcpy(&previousMap->valueChanges[0], dev_previousMapValueChanges, previousMapSize * sizeof(float), hipMemcpyDeviceToHost);
    
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "ConnectedMap::CalculateChanges Changes Extract failed: %s\n", hipGetErrorString(cudaStatus));
        }
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ConnectedMap::CalculateChanges Changes failed: %s\n", hipGetErrorString(cudaStatus));
    }

    hipMemcpy(&bias.weightChanges[0], dev_biasChanges, mapSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&weights.weightChanges[0], dev_weightChanges, mapSize * previousMapSize * previousMaps.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_threadOffset);

    hipFree(dev_activations);
    hipFree(dev_valueChanges);

    hipFree(dev_previousMapActivations);
    hipFree(dev_previousMapValues);
    hipFree(dev_previousMapValueChanges);
    hipFree(dev_weights);
    hipFree(dev_weightChanges);

    hipFree(dev_previousMapIndex);
    hipFree(dev_previousMapSize);
    hipFree(dev_bias);
    hipFree(dev_biasChanges);

    hipFree(dev_mapSize);
    hipFree(dev_previousMapCount);

    hipFree(dev_pooling);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ConnectedMap::CalculateChanges Free failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

void ConnectedMap::Improve()
{
    bias.Improve();
    weights.Improve();

    for(int i = 0; i < mapSize; i ++)
    {
        valueChanges[i] = 0;
    }
}

NeuralLayer::NeuralLayer()
{
}

NeuralLayer::NeuralLayer(int i_mapCount, int i_reduction, MapType i_type, NeuralLayer* i_previousLayer)
{
    id = ID++;

    layerType = i_type;

    mapCount = i_mapCount;

    if(layerType == Input)
    {
        inputMaps = vector<InputMap> (mapCount);

        for(int i = 0; i < mapCount; i ++)
        {
            inputMaps[i] = InputMap(i_reduction);
            mapPointer.push_back(&inputMaps[i]);
        }
    }
    else if(layerType == Convolutional)
    {
        convolutionalMaps = vector<ConvolutionalMap> (mapCount);

        for(int i = 0; i < mapCount; i ++)
        {
            convolutionalMaps[i] = ConvolutionalMap(i_reduction, i_previousLayer->mapPointer);
            mapPointer.push_back(&convolutionalMaps[i]);
        }
    }
    else if(layerType == Pooling)
    {
        poolingMaps = vector<PoolingMap> (mapCount);

        for(int i = 0; i < mapCount; i ++)
        {
            poolingMaps[i] = PoolingMap(i_reduction, i_previousLayer->mapPointer[i]);
            mapPointer.push_back(&poolingMaps[i]);
        }
    }
    else if(layerType == Connected)
    {
        connectedMaps = vector<ConnectedMap> (mapCount);

        for(int i = 0; i < mapCount; i ++)
        {
            connectedMaps[i] = ConnectedMap(i_reduction, i_previousLayer->mapPointer[i]);
            mapPointer.push_back(&connectedMaps[i]);
        }
    }
}

void NeuralLayer::SetValues(vector<vector<float>*> i_values)
{
    for(int i = 0; i < mapCount; i ++)
    {
        mapPointer[i]->SetValues(i_values[i]);
    }
}

vector<vector<float>*> NeuralLayer::GetValues()
{
    vector<vector<float>*> results;
    for(int i = 0; i < mapCount; i ++)
    {
        results.push_back(mapPointer[i]->GetValues());
    }
    return results;
}

void NeuralLayer::Correct(vector<float>* i_values)
{    
    mapPointer[0]->Correct(i_values);  
}

void NeuralLayer::PrintValues()
{
    cout << "Neural Layer (id: " << id << ")" << endl;
    for(int i = 0; i < mapCount; i ++)
    {
        mapPointer[i]->PrintValues();
    }
}

void NeuralLayer::Update()
{
    for(int i = 0; i < mapCount; i ++)
    {
        //cout << "       Update map " << (i + 1) << " / " << mapCount << endl;
        mapPointer[i]->Update();
    }
}

void NeuralLayer::CalculateChanges()
{
    for(int i = 0; i < mapCount; i ++)
    {
        //cout << "       Calculate map " << (i + 1) << " / " << mapCount << endl;
        mapPointer[i]->CalculateChanges();
    }
}

void NeuralLayer::Improve()
{
    for(int i = 0; i < mapCount; i ++)
    {
        mapPointer[i]->Improve();
    }
}

ConvolutionalNetwork::ConvolutionalNetwork(vector<MapType> i_layerType, vector<int> i_mapCount, vector<int> i_reduction)
{
    id = ID++;

    layerCount = i_layerType.size();

    layers = vector<NeuralLayer> (layerCount);

    for(int i = 0; i < layerCount; i ++)
    {
        layers[i] = NeuralLayer(i_mapCount[i], i_reduction[i], i_layerType[i], &layers[max(i - 1,0)]);
    }
}


void ConvolutionalNetwork::SetValues(vector<vector<float>*> i_values)
{
    layers[0].SetValues(i_values);
}

vector<vector<float>*> ConvolutionalNetwork::GetValues(int i_layer)
{
    return layers[i_layer].GetValues();
}

void ConvolutionalNetwork::Correct(vector<float>* i_values)
{    
    layers[layerCount - 1].Correct(i_values);  
}

void ConvolutionalNetwork::Train(vector<vector<float>*> i_trainValues, vector<float>* i_correctvalues)
{    
    //cout << "Set net input" << endl;
    SetValues(i_trainValues);

    //cout << "Update net" << endl;
    Update();

    //cout << "Get correct output" << endl;
    Correct(i_correctvalues);

    //cout << "Calculate net changes" << endl;
    CalculateChanges();
}

void ConvolutionalNetwork::PrintValues()
{
    cout << "Convolutional Network (id: " << id << ")" << endl;
    for(int i = 0; i < layerCount; i ++)
    {
        layers[i].PrintValues();
    }
}

void ConvolutionalNetwork::Update()
{
    for(int i = 0; i < layerCount; i ++)
    {
        //cout << "   Update layer " << (i + 1) << " / " << layerCount << endl;
        layers[i].Update();
    }
}

void ConvolutionalNetwork::CalculateChanges()
{
    for(int i = layerCount - 1; i >= 0; i --)
    {
        //cout << "   Calculate layer " << (i + 1) << " / " << layerCount << endl;
        layers[i].CalculateChanges();
    }
}

void ConvolutionalNetwork::Improve()
{
    for(int i = 0; i < layerCount; i ++)
    {
        layers[i].Improve();
    }
}