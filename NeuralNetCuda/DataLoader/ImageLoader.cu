#include "hip/hip_runtime.h"
#include "ImageLoader.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

vector<string> splitString(string i_text, string i_delimiter)
{
    string text = i_text;

    vector<string> textList;

    size_t pos = 0;

    while ((pos = text.find(i_delimiter)) != string::npos) {
        string subText = text.substr(0, pos);
        textList.push_back(subText);
        text.erase(0, pos + i_delimiter.length());
    }
    textList.push_back(text);

    return textList;
}

string removeChracters(string i_text, vector<char> i_characters)
{
    string text;

    for(int i = 0; i < i_text.size(); i ++)
    {
        bool isInCharacterList = false;
        for(int j = 0; j < i_characters.size(); j ++)
        {
            if(i_text[i] == i_characters[j])
            {
                isInCharacterList = true;
            }
        }

        if(!isInCharacterList)
        {
            text += i_text[i];
        }
    }

    return text;
}

float clip(float value, float minValue, float maxValue)
{
    return min(max(value, minValue), maxValue);
}

BoundingBox::BoundingBox()
{
}

BoundingBox::BoundingBox(int i_minX, int i_maxX, int i_minY, int i_maxY)
{
    minX = i_minX;
    maxX = i_maxX;
    minY = i_minY;
    maxY = i_maxY;
}

int BoundingBox::getAreaOfIntersection(BoundingBox* i_boundingBox)
{
    int xOverlap = max(min(maxX, i_boundingBox->maxX) - max(minX, i_boundingBox->minX), 0);
    int yOverlap = max(min(maxY, i_boundingBox->maxY) - max(minY, i_boundingBox->minY), 0);

    return xOverlap * yOverlap;
}

int BoundingBox::getCombinedArea(BoundingBox* i_boundingBox, int i_areaOfOverlap)
{
    int area = (maxX - minX) * (maxY - minY);
    int boundingBoxArea = (i_boundingBox->maxX - i_boundingBox->minX) * (i_boundingBox->maxY - i_boundingBox->minY);

    return area + boundingBoxArea - i_areaOfOverlap;
}

int BoundingBox::getSpanArea(BoundingBox* i_boundingBox)
{
    int width = max(maxX, i_boundingBox->maxX) - min(minX, i_boundingBox->minX);
    int heigth = max(maxY, i_boundingBox->maxY) - min(minY, i_boundingBox->minY);   

    return width * heigth; 
}

float BoundingBox::getRatioOfIntersection(BoundingBox* i_boundingBox)
{
    int areaOfIntersection = getAreaOfIntersection(i_boundingBox);
    int combinedArea = getCombinedArea(i_boundingBox, areaOfIntersection);
    int spanArea = getSpanArea(i_boundingBox);

    float intersectionOverUnion = (float) areaOfIntersection / (float) combinedArea;
    float zeroCorrector = ((float) (spanArea - combinedArea)) / (float) spanArea;

    return intersectionOverUnion - zeroCorrector;
}

bool BoundingBox::isIn(int x, int y)
{
    if (x < minX)
        return false;
    if (x > maxX)
        return false;
    if (y < minY)
        return false;
    if (y > maxY)
        return false;

    return true;
}

bool BoundingBox::isOnBounds(int x, int y)
{
    if(x < minX)
        return false;
    if(x > maxX)
        return false;
    if(y < minY)
        return false;
    if(y > maxY)
        return false;

    if(x != minX && x != maxX && y != minY && y != maxY)
        return false;

    return true;
}

BoundingBoxStorage::BoundingBoxStorage()
{
}

void BoundingBoxStorage::getAllBoundingBoxes(int i_minSize, int i_maxSize, float i_maxRatio, int i_imageSize, int i_skip, int i_sizeSkip)
{
    for(int x = 0; x < i_imageSize; x += i_skip)
    {
        for(int y = 0; y < i_imageSize; y += i_skip)
        {
            for(int w = i_minSize; w < i_maxSize; w += i_sizeSkip)
            {
                if(x + w >= i_imageSize)
                {
                    break;
                }

                for(int h = i_minSize; h < i_maxSize; h += i_sizeSkip)
                {
                    if(y + h >= i_imageSize)
                    {
                        break;
                    }

                    float ratio = (float) max(w, h) / (float) min(w, h);

                    if(ratio > i_maxRatio)
                    {
                        continue;
                    }
                    allBoundingBoxes.push_back(BoundingBox(x, x + w, y, y + h));
                }
            }
        }
    }
}

TrafficSign::TrafficSign()
{
}

TrafficSign::TrafficSign(string i_label, int i_minX, int i_minY, int i_maxX, int i_maxY)
{
    label = i_label;
    minX = i_minX;
    minY = i_minY;
    maxX = i_maxX;
    maxY = i_maxY;
}

void TrafficSign::printTrafficSign()
{
    cout << "Traffic sign: (label: " << label << ", bounding box: [" << minX << " - " << maxX << ", " << minY << " - " << maxY << "])" << endl;
}

Annotation::Annotation()
{
}

void Annotation::loadAnnotation(string i_fileName)
{
    fstream newfile;

    newfile.open(i_fileName,ios::in);
    if (newfile.is_open())
    {
        string line;

        TrafficSign* trafficSign = NULL;

        while(getline(newfile, line))
        {
            line = removeChracters(line, {' ', ',', '"'});

            vector<string> words = splitString(line, ":");

            if(words[0] == "width")
            {
                annotationWidth = stof(words[1]);
                continue;
            }
            
            if(words[0] == "height")
            {
                annotationHeight = stof(words[1]);
                continue;
            }
            
            if(words[0] == "label")
            {
                trafficSigns.push_back(TrafficSign());
                trafficSign = &trafficSigns[trafficSigns.size() - 1];

                trafficSign->label = words[1];
                continue;
            }

            if (trafficSign == NULL)
            {
                continue;
            }

            if(words[0] == "xmin")
            {
                trafficSign->minX = (stof(words[1]));
                continue;
            }
            
            if(words[0] == "xmax")
            {
                trafficSign->maxX = (stof(words[1]));
                continue;
            }
            
            if(words[0] == "ymin")
            {
                trafficSign->minY = (stof(words[1]));
                continue;
            }
            
            if(words[0] == "ymax")
            {
                trafficSign->maxY = (stof(words[1]));

                if(trafficSign->maxY - trafficSign->minY < 8)
                {
                    trafficSigns.pop_back();
                }
                else if(trafficSign->maxX - trafficSign->minX < 8)
                {
                    trafficSigns.pop_back();
                }
                continue;
            }
        }
        newfile.close();
    }

}

void Annotation::setTransformedAnnotation(vector<TrafficSign>* i_trafficSigns, int i_x, int i_y, int i_imageWidth, int i_imageHeight)
{
    for (int i = 0; i < i_trafficSigns->size(); i++)
    {
        TrafficSign* addedSign = &trafficSigns[i];

        float factorX = i_imageWidth / imageWidth;
        float factorY = i_imageHeight / imageHeight;

        float newMinX = (addedSign->minX - i_x) * factorX;
        float newMinY = (addedSign->minY - i_y) * factorY;
        float newMaxX = (addedSign->maxX - i_x) * factorX;
        float newMaxY = (addedSign->maxY - i_y) * factorY;

        trafficSigns.push_back(TrafficSign(addedSign->label, newMinX, newMinY, newMaxX, newMaxY));
    }
}

vector<float>* Annotation::getBoundingBoxRating(BoundingBoxStorage* i_allBoundingBoxes)
{
    ratings = vector<float> (0);

    float delta = 0.1f;

    for(int i = 0; i < i_allBoundingBoxes->allBoundingBoxes.size(); i ++)
    {
        float bestRating = -1;

        for(int j = 0; j < trafficSigns.size(); j ++)
        {
            float rating = i_allBoundingBoxes->allBoundingBoxes[i].getRatioOfIntersection(&trafficSigns[j]);

            if(rating >= trafficSigns[j].maxOverlap - delta)
            {
                bestRating = 1;
                break;
            }

            if(rating >= bestRating)
            {
                bestRating = rating;
            }
        }

        if(bestRating > 0.5f)
        {
            ratings.push_back(1);
        }
        else if(bestRating <= 0.0f && rand() % 100 > 50)
        {
            ratings.push_back(0);
        }
        else
        {
            ratings.push_back(-1); 
        }

    }

    return &ratings;
}

vector<float>* Annotation::getPixelRating(int i_inputSize, int i_outputSize)
{
    ratings = vector<float>(0);

    for (int y = 0; y < i_outputSize; y++)
    {
        for (int x = 0; x < i_outputSize; x++)
        {
            float rX = (float) x / (float) i_outputSize;
            float rY = (float) y / (float) i_outputSize;

            int nX = rX * i_inputSize;
            int nY = rY * i_inputSize;

            bool isIn = false;

            for (int i = 0; i < trafficSigns.size(); i++)
            {
                TrafficSign* trafficSign = &trafficSigns[i];
                if (trafficSign->isIn(nX, nY))
                {
                    isIn = true;
                    break;
                }
            }

            ratings.push_back(isIn ? 1 : 0);
        }
    }

    return &ratings;
}

void Annotation::printAnnotation()
{
    cout << "Annotation" << endl;
    for(int i = 0; i < trafficSigns.size(); i ++)
    {
        trafficSigns[i].printTrafficSign();
    }
}

Frame::Frame()
{

}

Frame::Frame(int i_x, int i_y, int i_imageWidth, int i_imageHeight)
{
    x = i_x;
    y = i_y;
    imageWidth = i_imageWidth;
    imageHeight = i_imageHeight;
}

Image::Image()
{
    imageSize = 0;
}

Image::Image(string i_path, int i_imageSize)
{
    setImageSize(i_imageSize);

    string path = removeChracters(i_path, { '"' });

    vector<string> filePaths = splitString(path, "\\");

    for (int i = 0; i < filePaths.size() - 2; i++)
    {
        sourcePath += filePaths[i] + "\\";
    }

    fileName = filePaths[filePaths.size() - 1].substr(0, filePaths[filePaths.size() - 1].size() - 4);

    imagePath = sourcePath + "images\\" + fileName + ".jpg";

    loadAnnotation(sourcePath + "annotations\\" + fileName + ".json");
    resizeAnnotation(i_imageSize);

    if(trafficSigns.size() == 0)
    {
        return;
    }
}

void Image::setImageSize(int i_imageSize)
{
    imageWidth = i_imageSize;
    imageHeight = i_imageSize;

    imageSize = i_imageSize;
}

void Image::loadImage(int i_imageSize)
{
    const char* pathChars = imagePath.c_str();

    int bpp;

    int rawWidth;
    int rawHeight;

    uint8_t* rgb_image = stbi_load(pathChars, &rawWidth, &rawHeight, &bpp, 3);

    float factorX = (float)rawWidth / (float)i_imageSize;
    float factorY = (float)rawHeight / (float)i_imageSize;

    for(int y = 0; y < i_imageSize; y ++)
    {
        for(int x = 0; x < i_imageSize; x ++)
        {
            unsigned bytePerPixel = 3;
            unsigned char* pixelData = rgb_image + (int(x * factorX) + rawWidth * int(y * factorY)) * bytePerPixel;

            rChannel.push_back((float)pixelData[0] / 255.0f);
            gChannel.push_back((float)pixelData[1] / 255.0f);
            bChannel.push_back((float)pixelData[2] / 255.0f);
        }
    }

    stbi_image_free(rgb_image);
}

void Image::resizeImage(int i_imageSize)
{
    vector<float> rawRChannel = rChannel;
    vector<float> rawGChannel = gChannel;
    vector<float> rawBChannel = bChannel;

    rChannel = vector<float>(i_imageSize * i_imageSize);
    gChannel = vector<float>(i_imageSize * i_imageSize);
    bChannel = vector<float>(i_imageSize * i_imageSize);

    float factorX = (float)imageWidth / (float)i_imageSize;
    float factorY = (float)imageHeight / (float)i_imageSize;

    for (int y = 0; y < i_imageSize; y++)
    {
        for (int x = 0; x < i_imageSize; x++)
        {
            int newIndex = x + i_imageSize * y;
            int pixelIndex = int(x * factorX) + imageWidth * int(y * factorY);

            rChannel[newIndex] = rawRChannel[pixelIndex];
            gChannel[newIndex] = rawGChannel[pixelIndex];
            bChannel[newIndex] = rawBChannel[pixelIndex];
        }
    }

    resizeAnnotation(i_imageSize);
    setImageSize(i_imageSize);
}

void Image::resizeAnnotation(int i_imageSize)
{
    for (int i = 0; i < trafficSigns.size(); i++)
    {
        TrafficSign* addedSign = &trafficSigns[i];

        float factorX = (float)i_imageSize / (float)annotationWidth;
        float factorY = (float)i_imageSize / (float)annotationHeight;

        float newMinX = clip((addedSign->minX) * factorX, 0, i_imageSize);
        float newMinY = clip((addedSign->minY) * factorY, 0, i_imageSize);
        float newMaxX = clip((addedSign->maxX) * factorX, 0, i_imageSize);
        float newMaxY = clip((addedSign->maxY) * factorY, 0, i_imageSize);

        addedSign->minX = newMinX;
        addedSign->minY = newMinY;
        addedSign->maxX = newMaxX;
        addedSign->maxY = newMaxY;
    }

    annotationWidth = i_imageSize;
    annotationHeight = i_imageSize;
}

Image Image::getSubImage(Frame i_frame)
{
    Image outputImage;

    if (i_frame.x < 0 || i_frame.x + i_frame.imageWidth >= imageWidth || i_frame.y < 0 || i_frame.y + i_frame.imageHeight >= imageHeight)
    {
        cout << "Error reading coordinates" << endl;
        cout << i_frame.x << ", " << i_frame.y << ", " << i_frame.imageWidth << " / " << imageWidth << ", " << i_frame.imageHeight << " / " << imageHeight << endl;

        return outputImage;
    }

    outputImage.rChannel = vector<float>(i_frame.imageWidth * i_frame.imageHeight);
    outputImage.gChannel = vector<float>(i_frame.imageWidth * i_frame.imageHeight);
    outputImage.bChannel = vector<float>(i_frame.imageWidth * i_frame.imageHeight);

    for (int y = 0; y < i_frame.imageHeight; y++)
    {
        for (int x = 0; x < i_frame.imageWidth; x++)
        {
            int pixelIndex = (x + i_frame.x) + imageWidth * (y + i_frame.y);
            int outputIndex = x + i_frame.imageWidth * y;

            outputImage.rChannel[outputIndex] = rChannel[pixelIndex];
            outputImage.gChannel[outputIndex] = gChannel[pixelIndex];
            outputImage.bChannel[outputIndex] = bChannel[pixelIndex];
        }
    }

    for (int i = 0; i < trafficSigns.size(); i++)
    {
        TrafficSign* addedSign = &trafficSigns[i];

        float factorX = imageWidth / i_frame.imageWidth;
        float factorY = imageHeight / i_frame.imageHeight;

        float newMinX = clip((addedSign->minX - i_frame.x), 0, i_frame.imageWidth);
        float newMinY = clip((addedSign->minY - i_frame.y), 0, i_frame.imageHeight);
        float newMaxX = clip((addedSign->maxX - i_frame.x), 0, i_frame.imageWidth);
        float newMaxY = clip((addedSign->maxY - i_frame.y), 0, i_frame.imageHeight);

        outputImage.trafficSigns.push_back(TrafficSign(addedSign->label, newMinX, newMinY, newMaxX, newMaxY));
    }

    outputImage.sourcePath = sourcePath;
    outputImage.fileName = fileName;

    outputImage.imageWidth = i_frame.imageWidth;
    outputImage.imageHeight = i_frame.imageHeight;

    outputImage.imageSize = i_frame.imageWidth;

    int pixelOverlap = 0;

    for (int y = 0; y < outputImage.imageHeight; y++)
    {
        for (int x = 0; x < outputImage.imageWidth; x++)
        {
            int pixelIndex = x + outputImage.imageWidth * y;

            for (int i = 0; i < outputImage.trafficSigns.size(); i++)
            {
                if (outputImage.trafficSigns[i].isIn(x, y))
                {
                    pixelOverlap++;
                    break;
                }
            }
        }
    }

    outputImage.objectPercentage = (float)pixelOverlap / (float)(outputImage.imageWidth * outputImage.imageHeight);

    return outputImage;
}

Frame Image::getSubImageFrame(int i_x, int i_y, int i_imageWidth, int i_imageHeight)
{
    Frame outputFrame = Frame(i_x, i_y, i_imageWidth, i_imageHeight);

    if (i_x < 0 || i_x + i_imageWidth >= imageWidth || i_y < 0 || i_y + i_imageHeight >= imageWidth)
    {
        cout << "Error reading coordinates" << endl;
        cout << i_x << ", " << i_y << ", " << i_imageWidth << " / " << imageWidth << ", " << i_imageHeight << " / " << imageHeight << endl;

        return outputFrame;
    }

    float maxOverlap = 0;

    vector<TrafficSign> signs;

    for (int i = 0; i < trafficSigns.size(); i++)
    {
        TrafficSign* addedSign = &trafficSigns[i];

        int unionX = max(addedSign->maxX, i_x + i_imageWidth) - min(i_x, addedSign->minX);
        int unionY = max(addedSign->maxY, i_y + i_imageHeight) - min(i_y, addedSign->minY);
        int unionArea = unionX * unionY;

        int newMinX = clip((addedSign->minX - i_x), 0, i_imageWidth);
        int newMinY = clip((addedSign->minY - i_y), 0, i_imageHeight);
        int newMaxX = clip((addedSign->maxX - i_x), 0, i_imageWidth);
        int newMaxY = clip((addedSign->maxY - i_y), 0, i_imageHeight);

        int overlapX = min(addedSign->maxX, i_x + i_imageWidth) - max(addedSign->minX, i_x);
        int overlapY = min(addedSign->maxY, i_y + i_imageHeight) - max(addedSign->minY, i_y);

        if (overlapX <= 0 || overlapY <= 0)
        {
            continue;
        }

        float overlap = (float)(overlapX * overlapY) / (float)unionArea;

        if (overlap <= maxOverlap)
        {
            continue;
        }

        maxOverlap = overlap;
    }

    outputFrame.percentage = maxOverlap;
    
    return outputFrame;
}

vector<Image> Image::getSubImages(vector<Frame> i_frames)
{
    vector<Image> subImages;
    for (int i = 0; i < i_frames.size(); i++)
    {           
        subImages.push_back(getSubImage(i_frames[i]));
    }

    return subImages;
}

vector<Frame> Image::getSubImageFrames(vector<int> i_sizes, vector<float> i_ratios, int i_skipping)
{
    vector<Frame> subFrames;
    for (int y = 0; y < imageHeight; y += i_skipping)
    {
        for (int x = 0; x < imageWidth; x += i_skipping)
        {
            for (int i = 0; i < i_sizes.size(); i++)
            {
                for (int j = 0; j < i_ratios.size(); j++)
                {
                    int maxX = x + i_sizes[i];
                    int maxY = y + i_sizes[i] * i_ratios[j];

                    if (maxX >= imageWidth || maxY >= imageHeight)
                        continue;

                    subFrames.push_back(getSubImageFrame(x, y, i_sizes[i], i_sizes[i] * i_ratios[j]));
                }
            }
        }
    }

    return subFrames;
}

vector<vector<float>*> Image::getImageData()
{
    return {&rChannel, &gChannel, &bChannel};
}

void Image::setOptimalResults(BoundingBoxStorage* i_allBoundingBoxes)
{

    for(int j = 0; j < trafficSigns.size(); j ++)
    {
        float bestRating = -1;
        int bestIndex = 0;

        for(int i = 0; i < i_allBoundingBoxes->allBoundingBoxes.size(); i ++)
        {
            float rating = i_allBoundingBoxes->allBoundingBoxes[i].getRatioOfIntersection(&trafficSigns[j]);

            if(rating > bestRating)
            {
                bestRating = rating;
                bestIndex = i;
            }
        }

        optimalResults.push_back(&i_allBoundingBoxes->allBoundingBoxes[bestIndex]);

        trafficSigns[j].maxOverlap = bestRating;
    }
}

void Image::setBestNetResults(vector<float>* i_results)
{
    float bestResult = -10;
    int bestIndex = 0;

    for(int i = 0; i < i_results->size(); i ++)
    {
        if ((*i_results)[i] > bestResult)
        {
            bestIndex = i;
            bestResult = (*i_results)[i];
        }

        if((*i_results)[i] > 0.7)
        {
            bestNetResults.push_back(i);
        }
    }

    for (int y = 0; y < imageSize; y++)
    {
        for (int x = 0; x < imageSize; x++)
        {
            int oY = (float)y / (float)imageSize * 61;
            int oX = (float)x / (float)imageSize * 61;

            int oIndex = oX + 61 * oY;

            pixelOutput.push_back((*i_results)[oIndex]);
        }
    }

}


void Image::printImage(int i_index)
{
    uint8_t* pixels = new uint8_t[imageSize * imageSize * 3];

    int index = 0;

    for(int y = 0; y < imageSize; y ++)
    {
        for(int x = 0; x < imageSize; x ++)
        {
            int pixelIndex = x + imageSize * y;

            pixels[index ++] = rChannel[pixelIndex] * 255.0f;
            pixels[index ++] = gChannel[pixelIndex] * 255.0f;
            pixels[index ++] = bChannel[pixelIndex] * 255.0f;

            for(int i = 0; i < trafficSigns.size(); i ++)
            {
                if(trafficSigns[i].isOnBounds(x, y))
                {
                    pixels[index - 1] = 0;
                    pixels[index - 2] = 0;
                    pixels[index - 3] = 255.0f;
                }
            }
        }
    }


    for (int i = 0; i < pixelOutput.size(); i++)
    {
        float darknessFactor = max(pixelOutput[i], 0.1f);
        
        //pixels[i * 3 + 0] = (float)pixels[i * 3 + 0] * darknessFactor;
        //pixels[i * 3 + 1] = (float)pixels[i * 3 + 1] * darknessFactor;
        //pixels[i * 3 + 2] = (float)pixels[i * 3 + 2] * darknessFactor;
    }

    string outputPath = sourcePath + "results\\" + fileName + to_string(i_index) + ".jpg";

    const char* pathChars = outputPath.c_str();

    stbi_write_jpg(pathChars, imageSize, imageSize, 3, pixels, 100);
}