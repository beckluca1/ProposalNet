#include "hip/hip_runtime.h"
#include "ImageLoader.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

vector<string> splitString(string i_text, string i_delimiter)
{
    string text = i_text;

    vector<string> textList;

    size_t pos = 0;

    while ((pos = text.find(i_delimiter)) != string::npos) {
        string subText = text.substr(0, pos);
        textList.push_back(subText);
        text.erase(0, pos + i_delimiter.length());
    }
    textList.push_back(text);

    return textList;
}

string removeChracters(string i_text, vector<char> i_characters)
{
    string text;

    for(int i = 0; i < i_text.size(); i ++)
    {
        bool isInCharacterList = false;
        for(int j = 0; j < i_characters.size(); j ++)
        {
            if(i_text[i] == i_characters[j])
            {
                isInCharacterList = true;
            }
        }

        if(!isInCharacterList)
        {
            text += i_text[i];
        }
    }

    return text;
}

float clip(float value, float minValue, float maxValue)
{
    return min(max(value, minValue), maxValue);
}

BoundingBox::BoundingBox()
{
}

BoundingBox::BoundingBox(int i_minX, int i_maxX, int i_minY, int i_maxY)
{
    minX = i_minX;
    maxX = i_maxX;
    minY = i_minY;
    maxY = i_maxY;
}

int BoundingBox::getAreaOfIntersection(BoundingBox* i_boundingBox)
{
    int xOverlap = max(min(maxX, i_boundingBox->maxX) - max(minX, i_boundingBox->minX), 0);
    int yOverlap = max(min(maxY, i_boundingBox->maxY) - max(minY, i_boundingBox->minY), 0);

    return xOverlap * yOverlap;
}

int BoundingBox::getCombinedArea(BoundingBox* i_boundingBox, int i_areaOfOverlap)
{
    int area = (maxX - minX) * (maxY - minY);
    int boundingBoxArea = (i_boundingBox->maxX - i_boundingBox->minX) * (i_boundingBox->maxY - i_boundingBox->minY);

    return area + boundingBoxArea - i_areaOfOverlap;
}

int BoundingBox::getSpanArea(BoundingBox* i_boundingBox)
{
    int width = max(maxX, i_boundingBox->maxX) - min(minX, i_boundingBox->minX);
    int heigth = max(maxY, i_boundingBox->maxY) - min(minY, i_boundingBox->minY);   

    return width * heigth; 
}

float BoundingBox::getRatioOfIntersection(BoundingBox* i_boundingBox)
{
    int areaOfIntersection = getAreaOfIntersection(i_boundingBox);
    int combinedArea = getCombinedArea(i_boundingBox, areaOfIntersection);
    int spanArea = getSpanArea(i_boundingBox);

    float intersectionOverUnion = (float) areaOfIntersection / (float) combinedArea;
    float zeroCorrector = ((float) (spanArea - combinedArea)) / (float) spanArea;

    return intersectionOverUnion - zeroCorrector;
}

bool BoundingBox::isIn(int x, int y)
{
    if (x < minX)
        return false;
    if (x > maxX)
        return false;
    if (y < minY)
        return false;
    if (y > maxY)
        return false;

    return true;
}

bool BoundingBox::isOnBounds(int x, int y)
{
    if(x < minX)
        return false;
    if(x > maxX)
        return false;
    if(y < minY)
        return false;
    if(y > maxY)
        return false;

    if(x != minX && x != maxX && y != minY && y != maxY)
        return false;

    return true;
}

BoundingBoxStorage::BoundingBoxStorage()
{
}

void BoundingBoxStorage::getAllBoundingBoxes(int i_minSize, int i_maxSize, float i_maxRatio, int i_imageSize, int i_skip, int i_sizeSkip)
{
    for(int x = 0; x < i_imageSize; x += i_skip)
    {
        for(int y = 0; y < i_imageSize; y += i_skip)
        {
            for(int w = i_minSize; w < i_maxSize; w += i_sizeSkip)
            {
                if(x + w >= i_imageSize)
                {
                    break;
                }

                for(int h = i_minSize; h < i_maxSize; h += i_sizeSkip)
                {
                    if(y + h >= i_imageSize)
                    {
                        break;
                    }

                    float ratio = (float) max(w, h) / (float) min(w, h);

                    if(ratio > i_maxRatio)
                    {
                        continue;
                    }
                    allBoundingBoxes.push_back(BoundingBox(x, x + w, y, y + h));
                }
            }
        }
    }
}

TrafficSign::TrafficSign()
{
}

TrafficSign::TrafficSign(string i_label, int i_minX, int i_minY, int i_maxX, int i_maxY)
{
    label = i_label;
    minX = i_minX;
    minY = i_minY;
    maxX = i_maxX;
    maxY = i_maxY;
}

void TrafficSign::printTrafficSign()
{
    cout << "Traffic sign: (label: " << label << ", bounding box: [" << minX << " - " << maxX << ", " << minY << " - " << maxY << "])" << endl;
}

Annotation::Annotation()
{
}

void Annotation::loadAnnotation(string i_fileName)
{
    float width = 0;
    float height = 0;

    imageWidth = imageSize;
    imageHeight = imageSize;

    fstream newfile;

    newfile.open(i_fileName,ios::in);
    if (newfile.is_open())
    {
        string line;

        TrafficSign* trafficSign = NULL;

        while(getline(newfile, line))
        {
            line = removeChracters(line, {' ', ',', '"'});

            vector<string> words = splitString(line, ":");

            if(words[0] == "width")
            {
                width = stof(words[1]);
                continue;
            }
            
            if(words[0] == "height")
            {
                height = stof(words[1]);
                continue;
            }
            
            if(words[0] == "label")
            {
                trafficSigns.push_back(TrafficSign());
                trafficSign = &trafficSigns[trafficSigns.size() - 1];

                trafficSign->label = words[1];
                continue;
            }

            if (trafficSign == NULL)
            {
                continue;
            }

            if(words[0] == "xmin")
            {
                trafficSign->minX = (stof(words[1]));
                continue;
            }
            
            if(words[0] == "xmax")
            {
                trafficSign->maxX = (stof(words[1]));
                continue;
            }
            
            if(words[0] == "ymin")
            {
                trafficSign->minY = (stof(words[1]));
                continue;
            }
            
            if(words[0] == "ymax")
            {
                trafficSign->maxY = (stof(words[1]));

                if(trafficSign->maxY - trafficSign->minY < 8)
                {
                    trafficSigns.pop_back();
                }
                else if(trafficSign->maxX - trafficSign->minX < 8)
                {
                    trafficSigns.pop_back();
                }
                continue;
            }
        }
        newfile.close();
    }

    //cout << "loaded annotation" << endl;
}

void Annotation::setTransformedAnnotation(vector<TrafficSign>* i_trafficSigns, int i_x, int i_y, int i_imageWidth, int i_imageHeight)
{
    for (int i = 0; i < i_trafficSigns->size(); i++)
    {
        TrafficSign* addedSign = &trafficSigns[i];

        float factorX = i_imageWidth / imageWidth;
        float factorY = i_imageHeight / imageHeight;

        float newMinX = (addedSign->minX - i_x) * factorX;
        float newMinY = (addedSign->minY - i_y) * factorY;
        float newMaxX = (addedSign->maxX - i_x) * factorX;
        float newMaxY = (addedSign->maxY - i_y) * factorY;

        trafficSigns.push_back(TrafficSign(addedSign->label, newMinX, newMinY, newMaxX, newMaxY));
    }
}

vector<float>* Annotation::getBoundingBoxRating(BoundingBoxStorage* i_allBoundingBoxes)
{
    ratings = vector<float> (0);

    float delta = 0.1f;

    for(int i = 0; i < i_allBoundingBoxes->allBoundingBoxes.size(); i ++)
    {
        float bestRating = -1;

        for(int j = 0; j < trafficSigns.size(); j ++)
        {
            float rating = i_allBoundingBoxes->allBoundingBoxes[i].getRatioOfIntersection(&trafficSigns[j]);

            if(rating >= trafficSigns[j].maxOverlap - delta)
            {
                bestRating = 1;
                break;
            }

            if(rating >= bestRating)
            {
                bestRating = rating;
            }
        }

        if(bestRating > 0.5f)
        {
            ratings.push_back(1);
        }
        else if(bestRating <= 0.0f && rand() % 100 > 50)
        {
            ratings.push_back(0);
        }
        else
        {
            ratings.push_back(-1); 
        }

    }

    return &ratings;
}

vector<float>* Annotation::getPixelRating(int i_inputSize, int i_outputSize)
{
    ratings = vector<float>(0);

    for (int y = 0; y < i_outputSize; y++)
    {
        for (int x = 0; x < i_outputSize; x++)
        {
            float rX = (float) x / (float) i_outputSize;
            float rY = (float) y / (float) i_outputSize;

            int nX = rX * i_inputSize;
            int nY = rY * i_inputSize;

            bool isIn = false;

            for (int i = 0; i < trafficSigns.size(); i++)
            {
                TrafficSign* trafficSign = &trafficSigns[i];
                if (trafficSign->isIn(nX, nY))
                {
                    isIn = true;
                    break;
                }
            }

            ratings.push_back(isIn ? 1 : 0);
        }
    }

    return &ratings;
}

void Annotation::printAnnotation()
{
    cout << "Annotation" << endl;
    for(int i = 0; i < trafficSigns.size(); i ++)
    {
        trafficSigns[i].printTrafficSign();
    }
}

Image::Image()
{

}

Image::Image(string i_path, int i_imageSize)
{
    imageSize = i_imageSize;

    string path = removeChracters(i_path, { '"' });
    //cout << "Path: " << path << endl;

    vector<string> filePaths = splitString(path, "\\");

    for (int i = 0; i < filePaths.size() - 2; i++)
    {
        sourcePath += filePaths[i] + "\\";
    }

    fileName = filePaths[filePaths.size() - 1].substr(0, filePaths[filePaths.size() - 1].size() - 4);

    //cout << "Source: " << sourcePath << ", Image name: " << fileName << endl;

    string imagePath = sourcePath + "images\\" + fileName + ".jpg";

    //cout << "Image path: " << imagePath << endl;

    //cout << "Annotation path: " << sourcePath << "annotations\\" << fileName << ".json" << endl;

    loadAnnotation(sourcePath + "annotations\\" + fileName + ".json");

    if(trafficSigns.size() == 0)
    {
        cout << "No traffic signs found" << endl;
        return;
    }

    loadImage(imagePath);
    resizeImage(i_imageSize);
}

void Image::loadImage(string i_path)
{
    const char* pathChars = i_path.c_str();

    int bpp;

    uint8_t* rgb_image = stbi_load(pathChars, &imageWidth, &imageHeight, &bpp, 3);

    for(int y = 0; y < imageHeight; y ++)
    {
        for(int x = 0; x < imageWidth; x ++)
        {
            unsigned bytePerPixel = 3;
            unsigned char* pixelData = rgb_image + (x + imageWidth * y) * bytePerPixel;

            rChannel.push_back((float)pixelData[0] / 255.0f);
            gChannel.push_back((float)pixelData[1] / 255.0f);
            bChannel.push_back((float)pixelData[2] / 255.0f);
        }
    }

    stbi_image_free(rgb_image);
}

void Image::resizeImage(int i_imageSize)
{
    vector<float> rawRChannel = rChannel;
    vector<float> rawGChannel = gChannel;
    vector<float> rawBChannel = bChannel;

    float ratioX = (float)imageWidth / (float)i_imageSize;

    int gaussianKernelWidth = max(ratioX / 2, 1.0f);
    int gaussianKernelSizeX = gaussianKernelWidth * 2 + 1;
    float deviationX = max(((float)gaussianKernelWidth / 3.0f), 1.0f);
    float sqrDeviationX = deviationX * deviationX;

    vector<float> gaussianKernelX = vector<float>(gaussianKernelSizeX);

    float sum = 0;
    
    for (int dX = -gaussianKernelWidth; dX < gaussianKernelWidth + 1; dX++)
    {
        float sqrDistance = dX * dX;
        float gaussian = exp(-sqrDistance / (2.0f * sqrDeviationX)) / sqrt(2.0f * 3.142f * sqrDeviationX);
        sum += gaussian;

        int kernelIndex = dX + gaussianKernelWidth;
        gaussianKernelX[kernelIndex] = gaussian;
    }

    float ratioY = (float)imageHeight / (float)i_imageSize;

    int gaussianKernelHeight = max(ratioY / 2, 1.0f);
    int gaussianKernelSizeY = gaussianKernelHeight * 2 + 1;
    float deviationY = max(((float)gaussianKernelHeight / 3.0f), 1.0f);
    float sqrDeviationY = deviationY * deviationY;

    vector<float> gaussianKernelY = vector<float>(gaussianKernelSizeY);

    for (int dY = -gaussianKernelHeight; dY < gaussianKernelHeight + 1; dY++)
    {
        float sqrDistance = dY * dY;
        float gaussian = exp(-sqrDistance / (2.0f * sqrDeviationY)) / sqrt(2.0f * 3.142f * sqrDeviationY);
        sum += gaussian;

        int kernelIndex = dY + gaussianKernelHeight;
        gaussianKernelY[kernelIndex] = gaussian;
    }

    rChannel = vector<float> (i_imageSize * imageHeight);
    gChannel = vector<float> (i_imageSize * imageHeight);
    bChannel = vector<float> (i_imageSize * imageHeight);

    for(int y = 0; y < imageHeight; y ++)
    {
        for(int x = 0; x < i_imageSize; x ++)
        {
            int pixelIndex = x + i_imageSize * y;

            float rawX = ratioX * x;
            float rawY = y;

            for (int dX = -gaussianKernelWidth; dX < gaussianKernelWidth + 1; dX++)
            {
                int convolutedX = rawX + dX;

                if (convolutedX < 0 || convolutedX >= imageWidth)
                {
                    continue;
                }

                int convolutedIndex = convolutedX + imageWidth * rawY;

                float factor = gaussianKernelX[dX + gaussianKernelWidth];
                    
                rChannel[pixelIndex] += rawRChannel[convolutedIndex] * factor;
                gChannel[pixelIndex] += rawGChannel[convolutedIndex] * factor;
                bChannel[pixelIndex] += rawBChannel[convolutedIndex] * factor;
            }
        }
    }

    rawRChannel = rChannel;
    rawGChannel = gChannel;
    rawBChannel = bChannel;

    rChannel = vector<float>(i_imageSize * i_imageSize);
    gChannel = vector<float>(i_imageSize * i_imageSize);
    bChannel = vector<float>(i_imageSize * i_imageSize);

    for (int y = 0; y < i_imageSize; y++)
    {
        for (int x = 0; x < i_imageSize; x++)
        {
            int pixelIndex = x + i_imageSize * y;

            float rawX = x;
            float rawY = ratioY * y;

            for (int dY = -gaussianKernelHeight; dY < gaussianKernelHeight + 1; dY++)
            {
                int convolutedY = rawY + dY;

                if (convolutedY < 0 || convolutedY >= imageHeight)
                {
                    continue;
                }

                int convolutedIndex = rawX + i_imageSize * convolutedY;

                float factor = gaussianKernelY[dY + gaussianKernelHeight];

                rChannel[pixelIndex] += rawRChannel[convolutedIndex] * factor;
                gChannel[pixelIndex] += rawGChannel[convolutedIndex] * factor;
                bChannel[pixelIndex] += rawBChannel[convolutedIndex] * factor;
            }
        }
    }

    for (int i = 0; i < trafficSigns.size(); i++)
    {
        TrafficSign* addedSign = &trafficSigns[i];

        float factorX = (float)i_imageSize / (float)imageWidth;
        float factorY = (float)i_imageSize / (float)imageHeight;

        float newMinX = clip((addedSign->minX) * factorX, 0, i_imageSize);
        float newMinY = clip((addedSign->minY) * factorY, 0, i_imageSize);
        float newMaxX = clip((addedSign->maxX) * factorX, 0, i_imageSize);
        float newMaxY = clip((addedSign->maxY) * factorY, 0, i_imageSize);

        addedSign->minX = newMinX;
        addedSign->minY = newMinY;
        addedSign->maxX = newMaxX;
        addedSign->maxY = newMaxY;
    }

    imageWidth = i_imageSize;
    imageHeight = i_imageSize;

    imageSize = i_imageSize;
}

Image Image::getSubImage(int i_x, int i_y, int i_imageWidth, int i_imageHeight)
{
    Image outputImage;

    if (i_x < 0 || i_x + i_imageWidth >= imageWidth || i_y < 0 || i_y + i_imageHeight >= imageWidth)
    {
        cout << "Error reading coordinates" << endl;
        cout << i_x << ", " << i_y << ", " << i_imageWidth << " / " << imageWidth << ", " << i_imageHeight << " / " << imageHeight << endl;

        return outputImage;
    }

    outputImage.rChannel = vector<float>(i_imageWidth * i_imageHeight);
    outputImage.gChannel = vector<float>(i_imageWidth * i_imageHeight);
    outputImage.bChannel = vector<float>(i_imageWidth * i_imageHeight);

    for (int y = 0; y < i_imageHeight; y++)
    {
        for (int x = 0; x < i_imageWidth; x++)
        {
            int pixelIndex = (x + i_x) + imageWidth * (y + i_y);
            int outputIndex = x + i_imageWidth * y;

            outputImage.rChannel[outputIndex] = rChannel[pixelIndex];
            outputImage.gChannel[outputIndex] = gChannel[pixelIndex];
            outputImage.bChannel[outputIndex] = bChannel[pixelIndex];
        }
    }

    for (int i = 0; i < trafficSigns.size(); i++)
    {
        TrafficSign* addedSign = &trafficSigns[i];

        float factorX = imageWidth / i_imageWidth;
        float factorY = imageHeight / i_imageHeight;

        float newMinX = clip((addedSign->minX - i_x), 0, i_imageWidth);
        float newMinY = clip((addedSign->minY - i_y), 0, i_imageHeight);
        float newMaxX = clip((addedSign->maxX - i_x), 0, i_imageWidth);
        float newMaxY = clip((addedSign->maxY - i_y), 0, i_imageHeight);

        outputImage.trafficSigns.push_back(TrafficSign(addedSign->label, newMinX, newMinY, newMaxX, newMaxY));
    }

    outputImage.sourcePath = sourcePath;
    outputImage.fileName = fileName;

    outputImage.imageWidth = i_imageWidth;
    outputImage.imageHeight = i_imageHeight;

    outputImage.imageSize = i_imageWidth;

    int pixelOverlap = 0;

    for (int y = 0; y < outputImage.imageHeight; y++)
    {
        for (int x = 0; x < outputImage.imageWidth; x++)
        {
            int pixelIndex = x + outputImage.imageWidth * y;

            for (int i = 0; i < outputImage.trafficSigns.size(); i++)
            {
                if (outputImage.trafficSigns[i].isIn(x, y))
                {
                    pixelOverlap++;
                    break;
                }
            }
        }
    }

    outputImage.objectPercentage = (float)pixelOverlap / (float)(outputImage.imageWidth * outputImage.imageHeight);

    return outputImage;
}

vector<Image> Image::getSubImages(vector<int> i_sizes, vector<float> i_ratios, int i_skipping)
{
    vector<Image> subImages;
    for (int y = 0; y < imageHeight; y += i_skipping)
    {
        for (int x = 0; x < imageWidth; x += i_skipping)
        {
            for (int i = 0; i < i_sizes.size(); i++)
            {
                for (int j = 0; j < i_ratios.size(); j++)
                {
                    int maxX = x + i_sizes[i];
                    int maxY = y + i_sizes[i] * i_ratios[j];

                    if (maxX >= imageWidth || maxY >= imageHeight)
                        continue;

                    subImages.push_back(getSubImage(x, y, i_sizes[i], i_sizes[i] * i_ratios[j]));
                }
            }
        }
    }

    return subImages;
}

vector<vector<float>*> Image::getImageData()
{
    return {&rChannel, &gChannel, &bChannel};
}

void Image::setOptimalResults(BoundingBoxStorage* i_allBoundingBoxes)
{

    for(int j = 0; j < trafficSigns.size(); j ++)
    {
        float bestRating = -1;
        int bestIndex = 0;

        for(int i = 0; i < i_allBoundingBoxes->allBoundingBoxes.size(); i ++)
        {
            float rating = i_allBoundingBoxes->allBoundingBoxes[i].getRatioOfIntersection(&trafficSigns[j]);

            if(rating > bestRating)
            {
                bestRating = rating;
                bestIndex = i;
            }
        }

        optimalResults.push_back(&i_allBoundingBoxes->allBoundingBoxes[bestIndex]);

        trafficSigns[j].maxOverlap = bestRating;

        //cout << "Optimal bounding box " << bestIndex << " iou: " << bestRating << endl;
    }
}

void Image::setBestNetResults(vector<float>* i_results)
{
    float bestResult = -10;
    int bestIndex = 0;

    for(int i = 0; i < i_results->size(); i ++)
    {
        if ((*i_results)[i] > bestResult)
        {
            bestIndex = i;
            bestResult = (*i_results)[i];
        }

        if((*i_results)[i] > 0.7)
        {
            bestNetResults.push_back(i);
        }
    }

    cout << "Best index " << bestIndex << ", confidence: " << bestResult << endl;

    for (int y = 0; y < imageSize; y++)
    {
        for (int x = 0; x < imageSize; x++)
        {
            int oY = (float)y / (float)imageSize * 61;
            int oX = (float)x / (float)imageSize * 61;

            int oIndex = oX + 61 * oY;

            pixelOutput.push_back((*i_results)[oIndex]);
        }
    }

    cout << "Pixel area " << pixelOutput.size() << endl;

}


void Image::printImage()
{
    //cout << "Image: (width:" << imageWidth << ", height: " << imageHeight << ")" << endl;

    /*for(int y = 0; y < imageSize; y ++)
    {
        for(int x = 0; x < imageSize; x ++)
        {
            int pixelIndex = x + imageSize * y;

            float r = rChannel[pixelIndex];
            float g = gChannel[pixelIndex];
            float b = bChannel[pixelIndex];

            int outputIndex = (int) (10.0f * (max(r, max(g, b)) + min( r, min(g, b))) / 2.0f);

            vector<char> outputCharacters = {' ', '.', ':', '-', '=', '+', '*', '#', '%', '@'};

            cout << outputCharacters[outputIndex];

        }
        cout << endl;
    }*/

    uint8_t* pixels = new uint8_t[imageSize * imageSize * 3];

    int index = 0;

    for(int y = 0; y < imageSize; y ++)
    {
        for(int x = 0; x < imageSize; x ++)
        {
            int pixelIndex = x + imageSize * y;

            pixels[index ++] = rChannel[pixelIndex] * 255.0f;
            pixels[index ++] = gChannel[pixelIndex] * 255.0f;
            pixels[index ++] = bChannel[pixelIndex] * 255.0f;

            for(int i = 0; i < trafficSigns.size(); i ++)
            {
                if(trafficSigns[i].isOnBounds(x, y))
                {
                    pixels[index - 1] = 0;
                    pixels[index - 2] = 0;
                    pixels[index - 3] = 255.0f;
                }
            }
        }
    }


    for (int i = 0; i < pixelOutput.size(); i++)
    {
        float darknessFactor = max(pixelOutput[i], 0.1f);
        
        //pixels[i * 3 + 0] = (float)pixels[i * 3 + 0] * darknessFactor;
        //pixels[i * 3 + 1] = (float)pixels[i * 3 + 1] * darknessFactor;
        //pixels[i * 3 + 2] = (float)pixels[i * 3 + 2] * darknessFactor;
    }

    string imagePath = sourcePath + "results\\" + fileName + ".jpg";

    const char* pathChars = imagePath.c_str();

    stbi_write_jpg(pathChars, imageSize, imageSize, 3, pixels, 100);
}